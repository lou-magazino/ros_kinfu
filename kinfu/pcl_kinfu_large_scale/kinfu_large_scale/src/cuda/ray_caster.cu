#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */
#include "device.hpp"
//#include <boost/graph/buffer_concepts.hpp>


namespace pcl
{
  namespace device
  {
    namespace kinfuLS
    {

    /* ************************** */
      __device__ __forceinline__ float3
      float3_mul_elements(const float3 & a,const float3 & b)
      {
        return make_float3(a.x * b.x,a.y * b.y,a.z * b.z);
      }

      template <class StoreCondition,class StoreAction,class SearchCondition>
      struct RayCaster
      {
        enum { CTA_SIZE_X = 32, CTA_SIZE_Y = 8 };

        Mat33 Rcurr;
        float3 tcurr;

        float time_step;
        float3 volume_size;

        float3 cell_size;
        float3 cell_size_inv;
        int cols, rows;

        mutable SearchCondition search_condition;
        mutable StoreCondition store_condition;
        mutable StoreAction store_action;

        PtrStep<short2> volume;

        float min_range;

        Intr intr;

        mutable PtrStep<float> vmap;

        __device__ __forceinline__ float3
        get_ray_next (int x, int y) const
        {
          float3 ray_next;
          ray_next.x = (x - intr.cx) / intr.fx;
          ray_next.y = (y - intr.cy) / intr.fy;
          ray_next.z = 1;
          return ray_next;
        }

        __device__ __forceinline__ bool
        checkInds (const int3& g) const
        {
          return (g.x >= 0 && g.y >= 0 && g.z >= 0 && g.x < VOLUME_X && g.y < VOLUME_Y && g.z < VOLUME_Z);
        }

        __device__ __forceinline__ bool
        checkCoords (const float3 & g) const
        {
          return search_condition.Evaluate(g);
        }

        __device__ __forceinline__ float
        readTsdf (int x, int y, int z, const pcl::gpu::kinfuLS::tsdf_buffer & buffer) const
        {
          const short2* pos = &(volume.ptr (buffer.voxels_size.y * z + y)[x]);
          shift_tsdf_pointer(&pos, buffer);
          return unpack_tsdf (*pos);
        }

        __device__ __forceinline__ void
        readTsdf (int x, int y, int z, const pcl::gpu::kinfuLS::tsdf_buffer & buffer,float& tsdf, int& weight) const
        {
          const short2* pos = &(volume.ptr (buffer.voxels_size.y * z + y)[x]);
          shift_tsdf_pointer(&pos, buffer);
          unpack_tsdf (*pos,tsdf,weight);
        }

        __device__ __forceinline__ float3
        fromMetersToCells (const float3 & point) const
        {
          return float3_mul_elements(point,cell_size_inv);
        }

        __device__ __forceinline__ float3
        fromCellsToMeters (const float3 & point) const
        {
          return float3_mul_elements(point,cell_size);
        }

        __device__ __forceinline__ int3
        getVoxelFromPoint (float3 point) const
        {
          return getVoxelFromCell(fromMetersToCells(point));
        }

        __device__ __forceinline__ int3
        getVoxelFromCell (const float3 & cell) const
        {
          int vx = __float2int_rd (cell.x);        // round to negative infinity
          int vy = __float2int_rd (cell.y);
          int vz = __float2int_rd (cell.z);

          return make_int3 (vx, vy, vz);
        }

        __device__ __forceinline__ float
        interpolateTrilineary (const float3& point, const pcl::gpu::kinfuLS::tsdf_buffer & buffer) const
        {
          const float3 cell = fromMetersToCells (point);
          return interpolateTrilinearyFromCell (cell,buffer);
        }

        __device__ __forceinline__ float
        interpolateTrilinearyFromCell (const float3& cell, const pcl::gpu::kinfuLS::tsdf_buffer & buffer) const
        {
          int3 g = getVoxelFromCell (cell);

          if (g.x <= 0 || g.x >= buffer.voxels_size.x - 1)
            return numeric_limits<float>::quiet_NaN ();

          if (g.y <= 0 || g.y >= buffer.voxels_size.y - 1)
            return numeric_limits<float>::quiet_NaN ();

          if (g.z <= 0 || g.z >= buffer.voxels_size.z - 1)
            return numeric_limits<float>::quiet_NaN ();

  /*      //OLD CODE
          float vx = (g.x + 0.5f) * cell_size.x;
          float vy = (g.y + 0.5f) * cell_size.y;
          float vz = (g.z + 0.5f) * cell_size.z;

          g.x = (point.x < vx) ? (g.x - 1) : g.x;
          g.y = (point.y < vy) ? (g.y - 1) : g.y;
          g.z = (point.z < vz) ? (g.z - 1) : g.z;

          float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
          float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
          float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

          float res = readTsdf (g.x + 0, g.y + 0, g.z + 0, buffer) * (1 - a) * (1 - b) * (1 - c) +
                      readTsdf (g.x + 0, g.y + 0, g.z + 1, buffer) * (1 - a) * (1 - b) * c +
                      readTsdf (g.x + 0, g.y + 1, g.z + 0, buffer) * (1 - a) * b * (1 - c) +
                      readTsdf (g.x + 0, g.y + 1, g.z + 1, buffer) * (1 - a) * b * c +
                      readTsdf (g.x + 1, g.y + 0, g.z + 0, buffer) * a * (1 - b) * (1 - c) +
                      readTsdf (g.x + 1, g.y + 0, g.z + 1, buffer) * a * (1 - b) * c +
                      readTsdf (g.x + 1, g.y + 1, g.z + 0, buffer) * a * b * (1 - c) +
                      readTsdf (g.x + 1, g.y + 1, g.z + 1, buffer) * a * b * c;
  */
          //NEW CODE
          float a = cell.x - (g.x + 0.5f); if (a<0) { g.x--; a+=1.0f; };
          float b = cell.y - (g.y + 0.5f); if (b<0) { g.y--; b+=1.0f; };
          float c = cell.z - (g.z + 0.5f); if (c<0) { g.z--; c+=1.0f; };

          float res = (1 - a) * (
                                  (1 - b) * (
                                          readTsdf (g.x + 0, g.y + 0, g.z + 0, buffer) * (1 - c) +
                                          readTsdf (g.x + 0, g.y + 0, g.z + 1, buffer) *      c 
                                          )
                                  + b * (
                                          readTsdf (g.x + 0, g.y + 1, g.z + 0, buffer) * (1 - c) +
                                          readTsdf (g.x + 0, g.y + 1, g.z + 1, buffer) *      c  
                                          )
                                  )
                          + a * (
                                  (1 - b) * (
                                          readTsdf (g.x + 1, g.y + 0, g.z + 0, buffer) * (1 - c) +
                                          readTsdf (g.x + 1, g.y + 0, g.z + 1, buffer) *      c 
                                          )
                                  + b * (
                                          readTsdf (g.x + 1, g.y + 1, g.z + 0, buffer) * (1 - c) +
                                          readTsdf (g.x + 1, g.y + 1, g.z + 1, buffer) *      c 
                                          )
                                  )
                                          ;
          return res;
        }

        __device__ void find_min_max_time(float3 ray_org, float3 ray_dir, float3 box_max, float &tnear, float &tfar) const
        {
          const float3 box_min = make_float3(0.f, 0.f, 0.f);

          // compute intersection of ray with all six bbox planes
          float3 invR = make_float3(1.f/ray_dir.x, 1.f/ray_dir.y, 1.f/ray_dir.z);
          float3 tbot = float3_mul_elements(invR,box_min - ray_org);
          float3 ttop = float3_mul_elements(invR,box_max - ray_org);

          // re-order intersections to find smallest and largest on each axis
          float3 tmin = make_float3(fminf(ttop.x, tbot.x), fminf(ttop.y, tbot.y), fminf(ttop.z, tbot.z));
          float3 tmax = make_float3(fmaxf(ttop.x, tbot.x), fmaxf(ttop.y, tbot.y), fmaxf(ttop.z, tbot.z));

          // find the largest tmin and the smallest tmax
          tnear = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
          tfar  = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));
        }


        __device__ __forceinline__ void
        operator () (pcl::gpu::kinfuLS::tsdf_buffer buffer) const
        {
          int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
          int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

          if (x >= cols || y >= rows)
            return;

          store_action.Init(*this,x,y);

          const float3 ray_start = tcurr;
          float3 ray_dir = normalized (Rcurr * get_ray_next (x, y));

          //ensure that it isn't a degenerate case
          ray_dir.x = (ray_dir.x == 0.f) ? 1e-15 : ray_dir.x;
          ray_dir.y = (ray_dir.y == 0.f) ? 1e-15 : ray_dir.y;
          ray_dir.z = (ray_dir.z == 0.f) ? 1e-15 : ray_dir.z;

          // computer time when entry and exit volume
          float time_start_volume;
          float time_exit_volume;
          find_min_max_time(ray_start,ray_dir,volume_size,time_start_volume,time_exit_volume);

          const float min_dist = 0.f;         //in meters
          time_start_volume = fmax (time_start_volume, min_dist);
          if (time_start_volume >= time_exit_volume)
            return;
          time_exit_volume -= time_step;

          float time_curr = time_start_volume;
          int3 g = getVoxelFromPoint (ray_start + ray_dir * time_curr);
          g.x = max (0, min (g.x, buffer.voxels_size.x - 1));
          g.y = max (0, min (g.y, buffer.voxels_size.y - 1));
          g.z = max (0, min (g.z, buffer.voxels_size.z - 1));

          float tsdf;
          int weight;
          readTsdf (g.x, g.y, g.z, buffer, tsdf, weight);

          //infinite loop guard
          const float max_time = fmin(time_exit_volume,3.0 * (volume_size.x + volume_size.y + volume_size.z));

          float curr_time_step = time_step;

          for (; time_curr < max_time; time_curr += curr_time_step)
          {
            float tsdf_prev = tsdf;
            int weight_prev = weight;

            const float3 world_pt = ray_start + ray_dir * (time_curr + time_step);
            if (!checkCoords (world_pt))
              continue;

            g = getVoxelFromPoint (world_pt);
            if (!checkInds (g))
              return;

            readTsdf (g.x, g.y, g.z, buffer, tsdf, weight);

            {
              float new_time_step;
              if (store_condition.ChangeTimeStep(tsdf_prev,tsdf,weight_prev,weight,curr_time_step,new_time_step))
              {
                if (new_time_step > min(cell_size.x,min(cell_size.y,cell_size.z)))
                {
                  tsdf = tsdf_prev;
                  weight = weight_prev;

                  time_curr -= curr_time_step;
                  curr_time_step = new_time_step;
                  continue;
                }
              }
            }

            if (tsdf_prev < 0.f && tsdf > 0.f)
              return;

            const bool zero_crossing = store_condition.Evaluate(tsdf_prev,tsdf,weight_prev,weight);
            if (zero_crossing && time_curr < min_range)
              return;

            if (zero_crossing)
              break; // break out of the cycle here, so Stores will be executed in sync by all threads
          }

          const float3 world_pt_prev = ray_start + ray_dir * (time_curr);
          const float3 world_pt = ray_start + ray_dir * (time_curr + curr_time_step);

          store_action.Store(world_pt_prev,world_pt,g,tsdf,weight,time_curr,time_step,ray_start,ray_dir,
            *this,x,y,buffer);
        }
      };

      struct SphereSearchCondition
      {
        SphereSearchCondition () {}
        SphereSearchCondition (const float3 & c,const float & r): sphere_center(c), sphere_radius(r) {}

        __device__ __forceinline__ bool Evaluate (const float3 & pt) const
          {
          const float xx = (sphere_center.x - pt.x);
          const float yy = (sphere_center.y - pt.y);
          const float zz = (sphere_center.z - pt.z);
          return xx * xx + yy * yy + zz * zz < sphere_radius * sphere_radius;
          }

        float3 sphere_center;
        float sphere_radius;
      };

      struct BBoxSearchCondition
      {
        BBoxSearchCondition () {}
        BBoxSearchCondition (const float3 & m,const float3 & M): bbox_min(m), bbox_max(M) {}

        __device__ __forceinline__ bool Evaluate (const float3 & pt) const
          {
          return (pt.x >= bbox_min.x && pt.y >= bbox_min.y && pt.z >= bbox_min.z &&
            pt.x < bbox_max.x && pt.y < bbox_max.y && pt.z < bbox_max.z);
          }

        float3 bbox_min;
        float3 bbox_max;
      };

      struct TrueSearchCondition
      {
        __device__ __forceinline__ bool Evaluate (const float3 & /*pt*/) const {return true; }
      };

      struct ZeroCrossingStoreCondition
      {
        __device__ __forceinline__ bool Evaluate(float tsdf_prev,float tsdf_curr,int /*weight_prev*/,int /*weight_curr*/)
        {
          return tsdf_prev > 0.0f && tsdf_curr < 0.0f;
        }

        __device__ __forceinline__ bool ChangeTimeStep(float /*tsdf_prev*/,float /*tsdf*/,int /*weight_prev*/,int /*weight_curr*/,
          float /*time_step*/,float & /*new_time_step*/)
        {
          return false;
        }
      };

      struct NotEmptyStoreCondition
      {
        __device__ __forceinline__ bool Evaluate(float /*tsdf_prev*/,float tsdf_curr,int /*weight_prev*/,int weight_curr)
        {
          return tsdf_curr < 0.0f || weight_curr == 0;
        }

        __device__ __forceinline__ bool ChangeTimeStep(float /*tsdf_prev*/,float /*tsdf*/,int weight_prev,int weight_curr,
          float time_step,float & new_time_step)
        {
          if (weight_curr == 0 && weight_prev != 0)
          {
            new_time_step = time_step / 2.0;
            return true;
          }
          return false;
        }
      };

      struct InterpolatePointAndNormalStoreAction
      {
        template <class _RayCaster>
        __device__ __forceinline__ void Init(_RayCaster & parent,int x,int y)
        {
          parent.vmap.ptr (y)[x] = numeric_limits<float>::quiet_NaN ();
          nmap.ptr (y)[x] = numeric_limits<float>::quiet_NaN ();
        }

        template <class _RayCaster>
        __device__ __forceinline__ void Store(const float3 & world_pt_prev,const float3 & world_pt,
          const int3 & /*voxel_id*/,float /*tsdf*/,float weight,
          float time_curr,float time_step,const float3 & ray_start,const float3 & ray_dir,
          const _RayCaster & parent,int x,int y,pcl::gpu::kinfuLS::tsdf_buffer & buffer)
        {
          if (weight == 0)
            return;

          float step_correction = 0.5;

          float Ftdt = parent.interpolateTrilineary (world_pt, buffer);
          if (isnan (Ftdt))
            return;

          float Ft = parent.interpolateTrilineary (world_pt_prev, buffer);
          if (isnan (Ft))
            return;

          if (abs(Ftdt - Ft) > 0.1)
            step_correction = __fdividef(Ft,Ftdt - Ft);

          float Ts = time_curr - time_step * step_correction;

          float3 vetex_found = ray_start + ray_dir * Ts;

          parent.vmap.ptr (y       )[x] = vetex_found.x;
          parent.vmap.ptr (y + parent.rows)[x] = vetex_found.y;
          parent.vmap.ptr (y + 2 * parent.rows)[x] = vetex_found.z;

          float3 t;
          float3 n;

          t = vetex_found;
          t.x += parent.cell_size.x;
          float Fx1 = parent.interpolateTrilineary (t, buffer);

          t = vetex_found;
          t.x -= parent.cell_size.x;
          float Fx2 = parent.interpolateTrilineary (t, buffer);

          n.x = (Fx1 - Fx2);

          t = vetex_found;
          t.y += parent.cell_size.y;
          float Fy1 = parent.interpolateTrilineary (t, buffer);

          t = vetex_found;
          t.y -= parent.cell_size.y;
          float Fy2 = parent.interpolateTrilineary (t, buffer);

          n.y = (Fy1 - Fy2);

          t = vetex_found;
          t.z += parent.cell_size.z;
          float Fz1 = parent.interpolateTrilineary (t, buffer);

          t = vetex_found;
          t.z -= parent.cell_size.z;
          float Fz2 = parent.interpolateTrilineary (t, buffer);

          n.z = (Fz1 - Fz2);

          n = normalized (n);

          nmap.ptr (y       )[x] = n.x;
          nmap.ptr (y + parent.rows)[x] = n.y;
          nmap.ptr (y + 2 * parent.rows)[x] = n.z;
        }

        PtrStep<float> nmap;
      };

      struct NotEmptyIntensityStoreAction
      {
        template <class _RayCaster>
        __device__ __forceinline__ void Init(_RayCaster & parent,int x,int y)
        {
          parent.vmap.ptr (y)[x] = numeric_limits<float>::quiet_NaN ();
          umap.ptr (y)[x] = -1.0; // empty
        }

        template <class _RayCaster>
        __device__ __forceinline__ void Store(const float3 & /*world_pt_prev*/,const float3 & world_pt,
          const int3 & /*voxel_id*/,float /*tsdf*/,float weight,
          float /*time_curr*/,float /*time_step*/,const float3 & /*ray_start*/,const float3 & /*ray_dir*/,
          const _RayCaster & parent,int x,int y,pcl::gpu::kinfuLS::tsdf_buffer & /*buffer*/)
        {
          parent.vmap.ptr (y       )[x] = world_pt.x;
          parent.vmap.ptr (y + parent.rows)[x] = world_pt.y;
          parent.vmap.ptr (y + 2 * parent.rows)[x] = world_pt.z;

          if (weight == 0)
            umap.ptr(y)[x] = 0.0; // unknown
          else
            umap.ptr(y)[x] = 1.0; // occupied
        }

        PtrStep<float> umap; // intensity values
      };

      template <class StoreCondition,class StoreAction,class SearchCondition>
      __global__ void
      rayCastKernel (const RayCaster<StoreCondition,StoreAction,SearchCondition> rc, pcl::gpu::kinfuLS::tsdf_buffer buffer) {
        rc (buffer);
      }

      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      template <class StoreCondition,class StoreAction,class SearchCondition>
      void
      templatedRaycast (const Intr& intr, const Mat33& Rcurr, const float3& tcurr,
                            float tranc_dist, float min_range, const float3& volume_size,
                            const PtrStep<short2>& volume, const pcl::gpu::kinfuLS::tsdf_buffer* buffer,MapArr& vmap,
                            const StoreCondition & store_condition,const StoreAction & store_action,
                            const SearchCondition & search_condition)
      {
        RayCaster<StoreCondition,StoreAction,SearchCondition> rc;

        rc.Rcurr = Rcurr;
        rc.tcurr = tcurr;

        rc.time_step = tranc_dist * 0.8f;

        rc.volume_size = volume_size;

        rc.cell_size.x = volume_size.x / buffer->voxels_size.x;
        rc.cell_size.y = volume_size.y / buffer->voxels_size.y;
        rc.cell_size.z = volume_size.z / buffer->voxels_size.z;
        rc.cell_size_inv.x = 1.0 / rc.cell_size.x;
        rc.cell_size_inv.y = 1.0 / rc.cell_size.y;
        rc.cell_size_inv.z = 1.0 / rc.cell_size.z;

        rc.cols = vmap.cols ();
        rc.rows = vmap.rows () / 3;

        rc.intr = intr;

        rc.min_range = min_range;

        rc.volume = volume;
        rc.vmap = vmap;

        rc.search_condition = search_condition;
        rc.store_action = store_action;
        rc.store_condition = store_condition;

        dim3 block (RayCaster<StoreCondition,StoreAction,SearchCondition>::CTA_SIZE_X,
          RayCaster<StoreCondition,StoreAction,SearchCondition>::CTA_SIZE_Y);
        dim3 grid (divUp (rc.cols, block.x), divUp (rc.rows, block.y));

        rayCastKernel<StoreCondition,StoreAction,SearchCondition><<<grid, block>>>(rc, *buffer);
        cudaSafeCall (hipGetLastError ());
        cudaSafeCall(hipDeviceSynchronize());
      }

      void
      raycast (const Intr& intr, const Mat33& Rcurr, const float3& tcurr,
                            float tranc_dist, float min_range, const float3& volume_size,
                            const PtrStep<short2>& volume, const pcl::gpu::kinfuLS::tsdf_buffer* buffer, MapArr& vmap, MapArr& nmap)
      {
        InterpolatePointAndNormalStoreAction ipan;
        ipan.nmap = nmap;
        templatedRaycast<ZeroCrossingStoreCondition,InterpolatePointAndNormalStoreAction,TrueSearchCondition>
          (intr,Rcurr,tcurr,tranc_dist,min_range,volume_size,volume,buffer,vmap,
          ZeroCrossingStoreCondition(),ipan,TrueSearchCondition());
      }

      void
      unkRaycast (const Intr& intr, const Mat33& Rcurr, const float3& tcurr,
                            float tranc_dist, float min_range, const float3& volume_size,
                            const PtrStep<short2>& volume, const pcl::gpu::kinfuLS::tsdf_buffer* buffer, MapArr& vmap, MapArr& umap)
      {
        NotEmptyIntensityStoreAction nesc;
        nesc.umap = umap;
        templatedRaycast<NotEmptyStoreCondition,NotEmptyIntensityStoreAction,TrueSearchCondition>(
          intr,Rcurr,tcurr,tranc_dist,min_range,volume_size,volume,buffer,vmap,
          NotEmptyStoreCondition(),nesc,TrueSearchCondition());
      }

      void
      unkRaycastBBox (const Intr& intr, const Mat33& Rcurr, const float3& tcurr,
                            float tranc_dist, float min_range, const float3& volume_size,
                            const PtrStep<short2>& volume, const pcl::gpu::kinfuLS::tsdf_buffer* buffer, MapArr& vmap, MapArr& umap,
                            const float3 & bbox_min,const float3 & bbox_max)
      {
        NotEmptyIntensityStoreAction nesc;
        nesc.umap = umap;
        templatedRaycast<NotEmptyStoreCondition,NotEmptyIntensityStoreAction,BBoxSearchCondition>
          (intr,Rcurr,tcurr,tranc_dist,min_range,volume_size,volume,buffer,vmap,
          NotEmptyStoreCondition(),nesc,BBoxSearchCondition(bbox_min,bbox_max));
      }
    }
  }
}
